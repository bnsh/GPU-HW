#include "hip/hip_runtime.h"
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
	//@@ Insert code to implement vector addition here
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < len) out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char ** argv) {
	wbArg_t args;
	int inputLength;
	float * hostInput1;
	float * hostInput2;
	float * hostOutput;
	float * deviceInput1;
	float * deviceInput2;
	float * deviceOutput;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *) malloc(inputLength * sizeof(float));
	hipMalloc(&deviceInput1, inputLength * sizeof(float));
	hipMalloc(&deviceInput2, inputLength * sizeof(float));
	hipMalloc(&deviceOutput, inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");


	wbSolution(args, hostOutput, inputLength);

	hipFree(deviceOutput); deviceOutput = NULL;
	hipFree(deviceInput2); deviceInput2 = NULL;
	hipFree(deviceInput1); deviceInput1 = NULL;
	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}

