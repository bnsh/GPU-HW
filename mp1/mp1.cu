#include "hip/hip_runtime.h"
// MP 1
#include <assert.h>
#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) out[idx] = in1[idx] + in2[idx];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    assert(hipMalloc(&deviceInput1, inputLength * sizeof(float)) == hipSuccess);
    assert(hipMalloc(&deviceInput2, inputLength * sizeof(float)) == hipSuccess);
    assert(hipMalloc(&deviceOutput, inputLength * sizeof(float)) == hipSuccess);


    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    assert(hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);


    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 blocksz(16,1,1);
    dim3 threadsz(16,1,1);


    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<<blocksz,threadsz>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    assert(hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    assert(hipFree(deviceInput1) == hipSuccess);
    assert(hipFree(deviceInput2) == hipSuccess);
    assert(hipFree(deviceOutput) == hipSuccess);


    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

