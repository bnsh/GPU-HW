#include "hip/hip_runtime.h"
#include	<wb.h>


#define wbCheck(stmt) do {													\
		hipError_t err = stmt;											   \
		if (err != hipSuccess) {											 \
			wbLog(ERROR, "Failed to run stmt ", #stmt);					   \
			wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));	\
			return -1;														\
		}																	 \
	} while(0)

const int BLOCKSZ = 32;
#define Mask_width  5
#define Mask_radius Mask_width/2

//@@ INSERT CODE HERE
__global__ void convolve(
	int imageWidth, int imageHeight, int imageChannels, const float *imageData,
	int maskRows, int maskColumns, const float *maskData,
	float *outputImageData
) {
/*
 * OK, so 
 */
	__shared__ float cpy[BLOCKSZ+2*Mask_radius][BLOCKSZ+2*Mask_radius]
}


int main(int argc, char* argv[]) {
	wbArg_t args;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char * inputImageFile;
	char * inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float * hostInputImageData;
	float * hostOutputImageData;
	float * hostMaskData;
	float * deviceInputImageData;
	float * deviceOutputImageData;
	float * deviceMaskData;

	args = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(args, 0);
	inputMaskFile = wbArg_getInputFile(args, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");


	wbTime_start(Copy, "Copying data to the GPU");
	hipMemcpy(deviceInputImageData,
			   hostInputImageData,
			   imageWidth * imageHeight * imageChannels * sizeof(float),
			   hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData,
			   hostMaskData,
			   maskRows * maskColumns * sizeof(float),
			   hipMemcpyHostToDevice);
	wbTime_stop(Copy, "Copying data to the GPU");


	wbTime_start(Compute, "Doing the computation on the GPU");
	//@@ INSERT CODE HERE
	dim3 blocksz(BLOCKSZ,BLOCKSZ,1);
	dim3 gridsz((((imageWidth-1)/blocksz.x)+1), (((imageHeight-1)/blocksz.y)+1),1);

	convolve<<<gridsz, blocksz>>>(
		imageWidth, imageHeight, imageChannels, deviceInputImageData,
		maskRows, maskColumns, deviceMaskData,
		deviceOutputImageData);

	wbTime_stop(Compute, "Doing the computation on the GPU");


	wbTime_start(Copy, "Copying data from the GPU");
	hipMemcpy(hostOutputImageData,
			   deviceOutputImageData,
			   imageWidth * imageHeight * imageChannels * sizeof(float),
			   hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(args, outputImage);

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);

	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
