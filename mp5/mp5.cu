#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {													\
	hipError_t err = stmt;											   \
	if (err != hipSuccess) {											 \
		wbLog(ERROR, "Failed to run stmt ", #stmt);					   \
		wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));	\
		return -1;														\
	}																	 \
} while(0)
	
__global__ void scan(float * input, float * output, int len) {
	//@@ Modify the body of this function to complete the functionality of
	//@@ the scan on the device
	//@@ You may need multiple kernel calls; write your kernels before this
	//@@ function and call them from here
	__shared__ float XY[BLOCK_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len) XY[i] = input[i];
	__syncthreads();

	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		// We start at 2 * stride - 1 and for each threadIdx.x we add a 2*stride
		int idx = (2 * stride - 1) + (threadIdx.x * stride * 2);
		if (idx < len) XY[idx] += XY[idx-stride];
		__syncthreads();
	}

	for (int stride = blockDim.x / 4; stride > 0; stride /= 2) {
		int idx = (3 * stride - 1) + (threadIdx.x * 2 * stride);
		if (idx < len) XY[idx] += XY[idx-stride];
		__syncthreads();
	}
	if (i < len) output[i] = XY[threadIdx.x];
}

int main(int argc, char ** argv) {
	wbArg_t args;
	float * hostInput; // The input 1D list
	float * hostOutput; // The output list
	float * deviceInput;
	float * deviceOutput;
	int numElements; // number of elements in the list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
	hostOutput = (float*) malloc(numElements * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ", numElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	wbCheck(hipMalloc((void**)&deviceInput, numElements*sizeof(float)));
	wbCheck(hipMalloc((void**)&deviceOutput, numElements*sizeof(float)));
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Clearing output memory.");
	wbCheck(hipMemset(deviceOutput, 0, numElements*sizeof(float)));
	wbTime_stop(GPU, "Clearing output memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	wbCheck(hipMemcpy(deviceInput, hostInput, numElements*sizeof(float), hipMemcpyHostToDevice));
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	//@@ Initialize the grid and block dimensions here
	dim3 blocksz(BLOCK_SIZE, 1, 1);
	dim3 gridsz((((numElements-1) / BLOCK_SIZE)+1), 1, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Modify this to complete the functionality of the scan
	//@@ on the deivce
	scan<<<gridsz, blocksz>>>(deviceInput, deviceOutput, numElements);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements*sizeof(float), hipMemcpyDeviceToHost));
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceInput);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, numElements);

	free(hostInput);
	free(hostOutput);

	return 0;
}

