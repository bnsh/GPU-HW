#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include	<wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {													\
		hipError_t err = stmt;											   \
		if (err != hipSuccess) {											 \
			wbLog(ERROR, "Failed to run stmt ", #stmt);					   \
			wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));	\
			return -1;														\
		}																	 \
	} while(0)

__global__ void total(float * input, float * output, int len) {
	//@@ Load a segment of the input vector into shared memory
	__shared__ float partial[2*BLOCK_SIZE];
	int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
	partial[2*threadIdx.x+0] = input[idx+0];
	partial[2*threadIdx.x+1] = input[idx+1];
	__syncthreads();

	//@@ Traverse the reduction tree
	for (int stride = blockDim.x; stride > 0; stride >>= 1) {
		if (threadIdx.x < stride) {
			int cidx = 2*blockIdx.x * blockDim.x + threadIdx.x + stride;
			if (cidx < len) partial[threadIdx.x] += partial[threadIdx.x + stride];
		}
		__syncthreads();
	}

	//@@ Write the computed sum of the block to the output vector at the 
	//@@ correct index
	if (threadIdx.x == 0) output[blockIdx.x] = partial[0];
}

int main(int argc, char ** argv) {
	int ii;
	wbArg_t args;
	float * hostInput; // The input 1D list
	float * hostOutput; // The output list
	float * deviceInput;
	float * deviceOutput;
	int numInputElements; // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numInputElements);

	numOutputElements = numInputElements / (BLOCK_SIZE<<1);
	if (numInputElements % (BLOCK_SIZE<<1)) {
		numOutputElements++;
	}
	hostOutput = (float*) malloc(numOutputElements * sizeof(float));

	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
	wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	//@@ Allocate GPU memory here
	wbCheck(hipMalloc(&deviceInput, sizeof(float) * numInputElements));
	wbCheck(hipMalloc(&deviceOutput, sizeof(float) * numOutputElements));

	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	//@@ Copy memory to the GPU here
	wbCheck(hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceOutput, hostOutput, numOutputElements * sizeof(float), hipMemcpyHostToDevice));

	wbTime_stop(GPU, "Copying input memory to the GPU.");
	//@@ Initialize the grid and block dimensions here
	dim3 blocksz(BLOCK_SIZE, 1, 1);
	dim3 gridsz(((numInputElements-1) / (2*BLOCK_SIZE))+1);

	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Launch the GPU Kernel here
	total<<<gridsz, blocksz>>>(deviceInput, deviceOutput, numInputElements);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	//@@ Copy the GPU memory back to the CPU here
	wbCheck(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

	wbTime_stop(Copy, "Copying output memory to the CPU");

	/********************************************************************
	 * Reduce output vector on the host
	 * NOTE: One could also perform the reduction of the output vector
	 * recursively and support any size input. For simplicity, we do not
	 * require that for this lab.
	 ********************************************************************/
	for (ii = 1; ii < numOutputElements; ii++) {
		hostOutput[0] += hostOutput[ii];
	}

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here
	wbCheck(hipFree(deviceInput));
	wbCheck(hipFree(deviceOutput));

	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, 1);

	free(hostInput);
	free(hostOutput);

	return 0;
}

