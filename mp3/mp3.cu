#include "hip/hip_runtime.h"

#include    <wb.h>

#define TILEWIDTH (16)
#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
			             int numARows, int numAColumns,
			             int numBRows, int numBColumns,
			             int numCRows, int numCColumns) {
	//@@ Insert code to implement matrix multiplication here
	//@@ You have to use shared memory for this MP
	__shared__ float mA[TILEWIDTH][TILEWIDTH];
	__shared__ float mB[TILEWIDTH][TILEWIDTH];
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
    hostC = (float *)(malloc(sizeof(float) * numCRows * numCColumns));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    wbCheck(hipMalloc(&deviceA, (sizeof(float) * numARows * numAColumns)));
    wbCheck(hipMalloc(&deviceB, (sizeof(float) * numBRows * numBColumns)));
    wbCheck(hipMalloc(&deviceC, (sizeof(float) * numCRows * numCColumns)));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    wbCheck(hipMemcpy(deviceA, hostA, (sizeof(float) * numARows * numAColumns), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceB, hostB, (sizeof(float) * numBRows * numBColumns), hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 blocksz(TILEWIDTH,TILEWIDTH,1);
    dim3 gridsz(((numCRows-1)/blocksz.x)+1,((numCColumns-1)/blocksz.y)+1,1);
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<blocksz, gridsz>>>(deviceA, deviceB, deviceC,
        numARows, numAColumns,
        numBRows, numBColumns,
        numCRows, numCColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    wbCheck(hipMemcpy(hostC, deviceC, (sizeof(float) * numCRows * numCColumns), hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    wbCheck(hipFree(deviceC)); deviceC = NULL;
    wbCheck(hipFree(deviceB)); deviceB = NULL;
    wbCheck(hipFree(deviceA)); deviceA = NULL;

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA); hostA = NULL;
    free(hostB); hostB = NULL;
    free(hostC); hostC = NULL;

    return 0;
}

