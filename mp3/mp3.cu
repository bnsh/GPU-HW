#include "hip/hip_runtime.h"
#include <sys/stat.h>
#include	<wb.h>

#define TILE_WIDTH (16)
#define wbCheck(stmt) do {													\
		hipError_t err = stmt;											   \
		if (err != hipSuccess) {											 \
			wbLog(ERROR, "Failed to run stmt ", #stmt);					   \
			wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));	\
			return -1;														\
		}																	 \
	} while(0)

// Compute C = A * B
__global__ void matrixMultiplyShared(const float * A, const float * B, float * C,
						 int numARows, int numAColumns,
						 int numBRows, int numBColumns,
						 int numCRows, int numCColumns) {
	//@@ Insert code to implement matrix multiplication here
	//@@ You have to use shared memory for this MP
	__shared__ float Atile[2 * TILE_WIDTH * TILE_WIDTH];
	float *Btile = Atile + TILE_WIDTH * TILE_WIDTH;

	int Ar = blockIdx.y * blockDim.y;
	int Bc = blockIdx.x * blockDim.x;

	float Cvalue = 0.0;
	for (int tile = 0; tile < (1+((numAColumns-1)/TILE_WIDTH)); ++tile) {
		int Ac = tile * TILE_WIDTH;
		int Aidx = (Ar+threadIdx.y) * numAColumns + (Ac + threadIdx.x);
		int Br = tile * TILE_WIDTH;
		int Bidx = (Br+threadIdx.y) * numBColumns + (Bc + threadIdx.x);

		Atile[threadIdx.y * TILE_WIDTH + threadIdx.x] = A[Aidx];
		Btile[threadIdx.y * TILE_WIDTH + threadIdx.x] = B[Bidx];
		__syncthreads();
		for (int i = 0; i < TILE_WIDTH; ++i) {
			// Interesting. An array out of bounds _READ_ causes memory faults.
			Cvalue += Atile[threadIdx.y * TILE_WIDTH + i] * Btile[i * TILE_WIDTH + threadIdx.x];
		}
		__syncthreads();
	}

	int Cr = Ar;
	int Cc = Bc;
	int Cidx = (Cr + threadIdx.y) * numCColumns + (Cc + threadIdx.x);
	C[Cidx] = Cvalue;
}

static float *myImport(const char *fn, int *rows, int *cols) __attribute__((unused));
static float *myImport(const char *fn, int *rows, int *cols) {
	float *rv = NULL;
	(*rows) = (*cols) = -1;
	struct stat buf;
	if (0 == stat(fn, &buf)) {
		char *rawdata = new char[buf.st_size+1]; memset(rawdata, '\0', buf.st_size+1);
		FILE *fp = fopen(fn, "r");
		if (fp) {
			assert((unsigned int)buf.st_size == fread(rawdata, 1, buf.st_size, fp));
			char *scrtch = NULL;
			int r = atoi(strtok_r(rawdata, " \t\r\n\f", &scrtch));
			int c = atoi(strtok_r(NULL, " \t\r\n\f", &scrtch));
			float *raw = (float *)malloc(sizeof(float) * r * c);
			for (int i = 0; i < r*c; ++i) raw[i] = atof(strtok_r(NULL, " \t\r\n\f", &scrtch));
			fclose(fp); fp = NULL;
			rv = raw;
			(*rows) = r;
			(*cols) = c;
		}
		delete[] rawdata; rawdata = NULL;
	}
	return rv;
}

int main(int argc, char ** argv) {
	wbArg_t args;
	float * hostA = NULL; // The A matrix
	float * hostB = NULL; // The B matrix
	float * hostC = NULL; // The output C matrix
	float * deviceA = NULL;
	float * deviceB = NULL;
	float * deviceC = NULL;
	int numARows; // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows; // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows; // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set this)

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostA = (float *) myImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
	hostB = (float *) myImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
	//@@ Set numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;
	//@@ Allocate the hostC matrix
	hostC = (float *)(malloc(sizeof(float) * numCRows * numCColumns));
	for (int i = 0; i < (numCRows * numCColumns); ++i) hostC[i] = 219 + i;
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
	wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
	wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

	wbTime_start(GPU, "Allocating GPU memory.");
	//@@ Allocate GPU memory here
	wbCheck(hipMalloc(&deviceA, (sizeof(float) * numARows * numAColumns)));
	wbCheck(hipMalloc(&deviceB, (sizeof(float) * numBRows * numBColumns)));
	wbCheck(hipMalloc(&deviceC, (sizeof(float) * numCRows * numCColumns)));

	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	//@@ Copy memory to the GPU here
	wbCheck(hipMemcpy(deviceA, hostA, (sizeof(float) * numARows * numAColumns), hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceB, hostB, (sizeof(float) * numBRows * numBColumns), hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceC, hostC, (sizeof(float) * numBRows * numBColumns), hipMemcpyHostToDevice));

	wbTime_stop(GPU, "Copying input memory to the GPU.");
	
	//@@ Initialize the grid and block dimensions here
	dim3 blocksz(TILE_WIDTH,TILE_WIDTH,1);
	dim3 gridsz(((numCColumns-1)/blocksz.x)+1,((numCRows-1)/blocksz.y)+1,1);
	
	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Launch the GPU Kernel here
	matrixMultiplyShared<<<gridsz, blocksz>>>(deviceA, deviceB, deviceC,
		numARows, numAColumns,
		numBRows, numBColumns,
		numCRows, numCColumns);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");
	
	wbTime_start(Copy, "Copying output memory to the CPU");
	//@@ Copy the GPU memory back to the CPU here
	wbCheck(hipMemcpy(hostC, deviceC, (sizeof(float) * numCRows * numCColumns), hipMemcpyDeviceToHost));

	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here
	wbCheck(hipFree(deviceC)); deviceC = NULL;
	wbCheck(hipFree(deviceB)); deviceB = NULL;
	wbCheck(hipFree(deviceA)); deviceA = NULL;

	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostC, numCRows, numCColumns);

	free(hostC); hostC = NULL;
	free(hostB); hostB = NULL;
	free(hostA); hostA = NULL;

	return 0;
}

